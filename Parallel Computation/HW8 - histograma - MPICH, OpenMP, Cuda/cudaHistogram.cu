#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdlib.h>
#include "myProto.h"


__global__  void calcBucketsKernel(int *arr, int part, int *bucket, int numOfBlocks, int numOfThreadsPerBlock) {
	int tid, blockId, start, end, bucketStart, i;

	tid = threadIdx.x;
	blockId = blockIdx.x;

	start = tid*part;
	end = start+part;

	if (tid < numOfThreadsPerBlock)
	{
		bucketStart = (blockId * numOfThreadsPerBlock + tid) * BUCKET_SIZE;
		for (i = start; i < end; ++i)
		{
			bucket[bucketStart + arr[i]]++;
		}
	}
}

__global__  void sumBucketsKernel(int *histogram, int *bucket, int numOfBlocks, int numOfThreadsPerBlock) {
	int tid, i, j;

	tid = threadIdx.x;

	for (i = 0; i < numOfBlocks; ++i)
	{
		for (j = 0; j < numOfThreadsPerBlock; ++j)
		{
			histogram[tid] += bucket[(i*numOfThreadsPerBlock + j)*BUCKET_SIZE+tid];\
		}
	}
}


int* histogramUsingCuda(int *arr, int arrSize) {

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    int threadsPerBlock, blocksPerGrid, *histogram;
    size_t size = arrSize * sizeof(int);

    threadsPerBlock = 32;
    blocksPerGrid = 2;

    // Allocate memory on GPU to copy the data from the host
    int *d_A;
    err = hipMalloc((void **)&d_A, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy data from host to the GPU memory
    err = hipMemcpy(d_A, arr, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

   // Allocate memory on GPU for each bucket
    int *bucket;
    err = hipMalloc((void **)&bucket, threadsPerBlock * blocksPerGrid * BUCKET_SIZE * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Reset bucket to 0 value
    err = hipMemset(bucket, 0, threadsPerBlock * blocksPerGrid * BUCKET_SIZE * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to reset allocated memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

   // Allocate memory on GPU for each bucket
    int *temp_histogram;
    err = hipMalloc((void **)&temp_histogram, BUCKET_SIZE * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Reset bucket to 0 value
    err = hipMemset(temp_histogram, 0, BUCKET_SIZE * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to reset allocated memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the calculation Kernel
    calcBucketsKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, arrSize/(threadsPerBlock*blocksPerGrid), bucket, blocksPerGrid, threadsPerBlock);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch calcBucketsKernel -  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error after launching calcBucketsKernel -  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the sum Kernel
    sumBucketsKernel<<<1, BUCKET_SIZE>>>(temp_histogram, bucket, blocksPerGrid, threadsPerBlock);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch sumBucketsKernel -  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error after launching sumBucketsKernel -  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the  result from GPU to the host memory.
    histogram = (int*) malloc(sizeof(int) * BUCKET_SIZE);
    if (!histogram)
    {
	fprintf(stderr, "Could not allocate array\n");
	exit(EXIT_FAILURE);	
    }
    err = hipMemcpy(histogram, temp_histogram, sizeof(int) * BUCKET_SIZE, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy result array from device to host -%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free allocated memory on GPU
    if (hipFree(d_A) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free allocated memory on GPU
    if (hipFree(bucket) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Free allocated memory on GPU
    if (hipFree(temp_histogram) != hipSuccess) {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return histogram;
}

