#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include "CudaFunc.h"
#include "OpenMPFunc.h"
#include "GeneralFunc.h"

__constant__ float cudaCompareTable[26][26];

__global__ void compareTwoSequences(char *cuda_seq1, char *cuda_seq2, double *cuda_results, int length, int offset, ScoreWeight weight)
{
	int tid, blockSize, blockId, index, i;
	double score;

	tid = threadIdx.x;
	blockSize = blockDim.x;
	blockId = blockIdx.x;
	index = ((blockId * blockSize) + tid);

	if (index < length)
	{
		score = 0;
		for (i = 0; i < length; i++)
		{
			if (!index || i < index)
			{
				score += cudaCompareTable[(int)(cuda_seq1[i + offset] - 'A')][(int)(cuda_seq2[i] - 'A')];
			}
			else if (i == index)
			{
				score -= weight.w4;
				i++;
			}
			else
			{
				score += cudaCompareTable[(int)(cuda_seq1[i + offset] - 'A')][(int)(cuda_seq2[i - 1] - 'A')];
			}
		}
		cuda_results[index] = score; // Each thread calculate a single offset and mutant combination score
	}
}

void *allocateDeviceMemory(int size)
{
	hipError_t err = hipSuccess;
	void *memory;
	err = hipMalloc((void **)&memory, size);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to allocate device memory for cuda_seq1 - %s\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	return memory;
}

void copyToMemory(void *to, void *from, int size, int dir)
{
	// dir = 1 -> Copy from host to the device
	// dir = 0 -> Copy from device to the host
	hipError_t err = hipSuccess;
	err = hipMemcpy(to, from, size, (dir ? hipMemcpyHostToDevice : hipMemcpyDeviceToHost));
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy from %s - %s\n", (dir ? "host to device" : "device to host"),
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void freeDeviceMemory(void *memory)
{
	hipError_t err = hipSuccess;
	if (hipFree(memory) != hipSuccess)
	{
		fprintf(stderr, "Failed to free allocated memory from device - %s\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

BestMatch findBestMatchingComparison(Sequence sequence1, Sequence sequence2, float *compareTable, int lowOffset, int highOffset, ScoreWeight weight)
{
	hipError_t err = hipSuccess;
	hipDeviceProp_t property;
	char *cuda_seq1, *cuda_seq2;
	double *cuda_results, *results;
	int threadsPerBlock, blocksPerGrid, i, j, resultLength, bestMutant;
	BestMatch bestMatch;

	// Initial settings
	bestMatch.offset = 0;
	bestMatch.mutantIndex = 0;
	bestMatch.score = -INFINITY;
	resultLength = sequence2.length + 1;

	/* 
	* Trying to use 1 block for each comparison for more use of cuda threads
	* Max threads per block: 1024
	* Max length of sequence2: 2000
	* Max blocks per sequence2: 2
	*/
	hipGetDeviceProperties(&property, 0);
	threadsPerBlock = (property.maxThreadsPerBlock > resultLength ? resultLength : property.maxThreadsPerBlock);
	blocksPerGrid = (resultLength / threadsPerBlock) + (resultLength % threadsPerBlock != 0);

	// Allocate memory on GPU device
	cuda_seq1 = (char*)allocateDeviceMemory(sequence1.length);
	cuda_seq2 = (char*)allocateDeviceMemory(sequence2.length);
	cuda_results = (double*)allocateDeviceMemory(resultLength*sizeof(double));

	// Copy the sequences from host to the GPU memory
	copyToMemory(cuda_seq1, sequence1.seq, sequence1.length, 1);
	copyToMemory(cuda_seq2, sequence2.seq, sequence2.length, 1);

	// Copy the compare table to cuda
	err = hipMemcpyToSymbol(HIP_SYMBOL(cudaCompareTable), compareTable, sizeof(float) * COMPARE_TABLE_SIZE * COMPARE_TABLE_SIZE);
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to copy compareTable from host to GPU memory - %s\n",
				hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// Allocate memory on host to copy the comparison result from the GPU
	results = (double *)malloc(sizeof(double) * resultLength);
	if (!results)
	{
		fprintf(stderr, "Failed to allocate results\n");
		exit(EXIT_FAILURE);
	}

	for (i = lowOffset; i < highOffset; i++)
	{
		// Compare two sequences
		compareTwoSequences<<<blocksPerGrid, threadsPerBlock>>>(cuda_seq1, cuda_seq2, cuda_results, resultLength - (!i ? 1 : 0), i, weight);

		// Copy result from the GPU to host memory
		err = hipMemcpy(results, cuda_results, resultLength*sizeof(double), hipMemcpyDeviceToHost);
		if (err != hipSuccess)
		{
			fprintf(stderr,
					"Failed to copy cuda_result from GPU to host memory - %s\n",
					hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		// Sending to OpenMP to calculate the score
		bestMutant = calculateScore(results, sequence2.length + (j > 0 ? 1 : 0));

		// Keeping the highest score details in a variable (bestMatch)
		if (results[bestMutant] > bestMatch.score)
		{
			bestMatch.offset = i;
			bestMatch.mutantIndex = bestMutant;
			bestMatch.score = results[bestMutant];
		}
	}

	// Free allocated memory on GPU device
	freeDeviceMemory(cuda_seq1);
	freeDeviceMemory(cuda_seq2);
	freeDeviceMemory(cuda_results);

	free(results);
	return bestMatch;
}
